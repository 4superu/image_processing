#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define IMAGE_SIZE 100
#define DET_SIZE 150

__global__ void multiDetection(float *cuda_detector, float *cuda_image){
  int i,j,angle,detector_coord;
  float x,y,s,pr;

  angle = blockIdx.x;
  i= blockIdx.y * blockDim.y + threadIdx.y;
  j= blockIdx.z * blockDim.z + threadIdx.z;

  x = (i+0.5) - (IMAGE_SIZE/2.0);
  y = (IMAGE_SIZE/2.0) - (j+0.5);

  s = x*cos(angle*M_PI/180.0) - y*sin(angle*M_PI/180.0);
  pr = s - floor(s);

  detector_coord = (int)(floor(s)+(DET_SIZE/2.0));

  atomicAdd(&cuda_detector[angle*DET_SIZE+detector_coord], cuda_image[j*IMAGE_SIZE+i]*(1.0-pr));
  atomicAdd(&cuda_detector[angle*DET_SIZE+detector_coord+1], cuda_image[j*IMAGE_SIZE+i]*pr);
}

__global__ void multiReconstruction(float *cuda_detector, float *cuda_rec_image){
  int i,j,angle,detector_coord;
  float x,y,s,pr;

  angle = blockIdx.x;
  i= blockIdx.y * blockDim.y + threadIdx.y;
  j= blockIdx.z * blockDim.z + threadIdx.z;

  x = (i+0.5) - (IMAGE_SIZE/2.0);
  y = (IMAGE_SIZE/2.0) - (j+0.5);

  s = x*cos(angle*M_PI/180.0) - y*sin(angle*M_PI/180.0);
  pr = s - floor(s);

  detector_coord = (int)(floor(s)+(DET_SIZE/2.0));

  atomicAdd(&cuda_rec_image[j*IMAGE_SIZE+i], cuda_detector[angle*DET_SIZE+detector_coord]*(1-pr));
  atomicAdd(&cuda_rec_image[j*IMAGE_SIZE+i], cuda_detector[angle*DET_SIZE+detector_coord+1]*pr);
}

int main(){
  float *image, *cuda_image;
  float *rec_image, *cuda_rec_image;
  float *detector, *cuda_detector;
  float time_ms = 0.0f;
  hipEvent_t start, stop;
  FILE *fp;

  image = (float*)malloc(sizeof(float)*(IMAGE_SIZE*IMAGE_SIZE));
  rec_image = (float*)calloc((IMAGE_SIZE*IMAGE_SIZE),sizeof(float));
  detector = (float*)calloc((DET_SIZE*360),sizeof(float));
  fp = fopen("./image/circle_100-100.raw", "rb");

  fread(image, sizeof(float), IMAGE_SIZE*IMAGE_SIZE, fp);
  fclose(fp);

  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipMalloc((void**)&cuda_detector, sizeof(float)*DET_SIZE*360);
  hipMalloc((void**)&cuda_image, sizeof(float)*IMAGE_SIZE*IMAGE_SIZE);
  hipMalloc((void**)&cuda_rec_image, sizeof(float)*IMAGE_SIZE*IMAGE_SIZE);

  hipMemcpy(cuda_image, image, sizeof(float)*IMAGE_SIZE*IMAGE_SIZE, hipMemcpyHostToDevice);
  hipMemcpy(cuda_detector, detector, sizeof(float)*DET_SIZE*360, hipMemcpyHostToDevice);
  hipMemcpy(cuda_rec_image, rec_image, sizeof(float)*IMAGE_SIZE*IMAGE_SIZE, hipMemcpyHostToDevice);

  hipEventRecord(start, 0);

  dim3 blocks(360,10,10);
  dim3 threads(1,10,10);

  multiDetection<<< blocks, threads >>>(cuda_detector, cuda_image);
  hipDeviceSynchronize();
  hipMemcpy(detector, cuda_detector, sizeof(float)*DET_SIZE*360, hipMemcpyDeviceToHost);

  multiReconstruction<<< blocks, threads >>>(cuda_detector, cuda_rec_image);
  hipDeviceSynchronize();
  hipMemcpy(rec_image, cuda_rec_image, sizeof(float)*IMAGE_SIZE*IMAGE_SIZE, hipMemcpyDeviceToHost);

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time_ms, start, stop);
  printf("time: %f ms\n",time_ms);
  hipEventDestroy(start);
  hipEventDestroy(stop);

  fp = fopen("./cuda_image/cuda_inter_circle_detection_150-360.raw", "wb");
  fwrite(detector, sizeof(float),DET_SIZE*360,fp);
  fclose(fp);

  fp = fopen("./cuda_image/cuda_inter_circle_100-100.raw", "wb");
  fwrite(rec_image, sizeof(float),IMAGE_SIZE*IMAGE_SIZE,fp);
  fclose(fp);

  free(image);
  free(rec_image);
  free(detector);

  hipFree(cuda_image);
  hipFree(cuda_detector);
  hipFree(cuda_rec_image);


}
